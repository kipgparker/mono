#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

template <typename scalar_t, typename cache_t>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key,    // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,  // [num_tokens, num_heads, head_size]
    cache_t* __restrict__ key_cache,     // [num_blocks, block_size, num_heads,
                                         // head_size]
    cache_t* __restrict__ value_cache,   // [num_blocks, block_size, num_heads,
                                         // head_size]
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int64_t block_stride, const int64_t page_stride,
    const int64_t head_stride, const int64_t key_stride,
    const int64_t value_stride, const int num_heads, const int head_size,
    const int block_size) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  // NOTE: slot_idx can be -1 if the token is padded
  if (slot_idx < 0) {
    return;
  }
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;
  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_key_idx = token_idx * key_stride + i;
    const int64_t src_value_idx = token_idx * value_stride + i;
    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int64_t tgt_key_value_idx = block_idx * block_stride +
                                      block_offset * page_stride +
                                      head_idx * head_stride + head_offset;
    key_cache[tgt_key_value_idx] = key[src_key_idx];
    value_cache[tgt_key_value_idx] = value[src_value_idx];
  }
}

#define CALL_RESHAPE_AND_CACHE_FLASH(KV_T, CACHE_T) \
  reshape_and_cache_flash_kernel<KV_T, CACHE_T> \
      <<<grid, block, 0, stream>>>( \
          reinterpret_cast<KV_T*>(key.data_ptr()), \
          reinterpret_cast<KV_T*>(value.data_ptr()), \
          reinterpret_cast<CACHE_T*>(key_cache.data_ptr()), \
          reinterpret_cast<CACHE_T*>(value_cache.data_ptr()), \
          slot_mapping.data_ptr<int64_t>(), block_stride, page_stride, \
          head_stride, key_stride, value_stride, num_heads, head_size, block_size);

#define DISPATCH_FLOAT_HALF(SRC_DTYPE, FN) \
  if (SRC_DTYPE == at::ScalarType::Float) { \
    FN(float, float); \
  } else if (SRC_DTYPE == at::ScalarType::Half) { \
    FN(at::Half, at::Half); \
  } else { \
    TORCH_CHECK(false, "Only float32 and float16 are supported."); \
  }

void reshape_and_cache_flash(
    torch::Tensor& key,        // [num_tokens, num_heads, head_size]
    torch::Tensor& value,      // [num_tokens, num_heads, head_size]
    torch::Tensor& key_cache,  // [num_blocks, block_size, num_heads, head_size]
    torch::Tensor&
        value_cache,  // [num_blocks, block_size, num_heads, head_size]
    torch::Tensor& slot_mapping) {  // [num_tokens] or [num_actual_tokens]
  // NOTE(woosuk): In vLLM V1, key.size(0) can be different from
  // slot_mapping.size(0) because of padding for CUDA graphs.
  // In vLLM V0, key.size(0) is always equal to slot_mapping.size(0) because
  // both include padding.
  // In vLLM V1, however, key.size(0) can be larger than slot_mapping.size(0)
  // since key includes padding for CUDA graphs, while slot_mapping does not.
  // In this case, slot_mapping.size(0) represents the actual number of tokens
  // before padding.
  // For compatibility with both cases, we use slot_mapping.size(0) as the
  // number of tokens.
  int num_tokens = slot_mapping.size(0);
  int num_heads = key.size(1);
  int head_size = key.size(2);
  int block_size = key_cache.size(1);

  int64_t key_stride = key.stride(0);
  int64_t value_stride = value.stride(0);
  int64_t block_stride = key_cache.stride(0);
  int64_t page_stride = key_cache.stride(1);
  int64_t head_stride = key_cache.stride(2);
  TORCH_CHECK(key_cache.stride(0) == value_cache.stride(0));

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(key));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_FLOAT_HALF(key.dtype(), CALL_RESHAPE_AND_CACHE_FLASH);
}